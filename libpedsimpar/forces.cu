#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_vector.h"
#include <thrust/device_vector.h>
#include <thrust/fill.h>

template<typename T>
struct line2col: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ line2col(T C) :
            C(C)
    {
    }

    __host__ __device__ T operator()(T i)
    {
        return i / C;
    }
};

__global__ void kernelLookahead(double2 *e, double2 *p, double2 *v, int* tmp, int N) {
    // compute lookahead counts

    int other = blockIdx.x * blockDim.x + threadIdx.x;
    int me = blockIdx.y * blockDim.y + threadIdx.y;

    if (me >= N || other >= N) {
        return;
    }

    if (me != other) {
        const double pi = 3.14159265;
        double distancex = p[other].x - p[me].x;
        double distancey = p[other].y - p[me].y;
        double dist2 = distancex*distancex + distancey*distancey;
        if (dist2 < 400) {
            double at2v = atan2(-e[me].x, -e[me].y);
            double at2d = atan2(-distancex, -distancey);
            double at2v2 = atan2(-v[other].x, -v[other].y);
            double s = at2d - at2v;
            if (s > pi) s -= 2*pi;
            if (s < -pi) s += 2*pi;
            double vv = at2v - at2v2;
            if (vv > pi) vv -= 2*pi;
            if (vv < -pi) vv += 2*pi;
            if (abs(vv) > 2.5) {
                if ((s < 0) && (s > -0.3))
                    tmp[me*N + other] = -1;
                if ((s > 0) && (s < 0.3))
                    tmp[me*N + other] = 1;
            }
        }
    }
}

void cudaLookaheadCount(vector<Ped::Tagent*> &agents, vector<int> &counts) {
    int N = agents.size();
    thrust::host_vector<double2> pvec_host(N);
    thrust::host_vector<double2> vvec_host(N);
    thrust::host_vector<double2> evec_host(N);

    thrust::device_vector<int> tmpvec(N*N);
    thrust::fill(tmpvec.begin(), tmpvec.end(), 0);

    int i = 0;
    for (std::vector<Ped::Tagent*>::iterator iter = agents.begin(); iter!=agents.end(); ++iter) {
        const Ped::Tagent* agent = *iter;
        Ped::Tvector op = agent->getPosition();
        Ped::Tvector ov = agent->getVelocity();
        Ped::Tvector oe = agent->getDesiredDirection();
        pvec_host[i] = make_double2(op.x, op.y);
        vvec_host[i] = make_double2(ov.x, ov.y);
        evec_host[i] = make_double2(oe.x, oe.y);
        i++;
    }

    thrust::device_vector<double2> pvec = pvec_host;
    thrust::device_vector<double2> vvec = vvec_host;
    thrust::device_vector<double2> evec = evec_host;

    dim3 blockDim(32, 32);
    dim3 gridDim((N + 32 - 1)/32, (N + 32 - 1)/32);
    kernelLookahead<<<gridDim, blockDim>>>(
        thrust::raw_pointer_cast(evec.data()),
        thrust::raw_pointer_cast(pvec.data()),
        thrust::raw_pointer_cast(vvec.data()),
        thrust::raw_pointer_cast(tmpvec.data()),
        N
    );

    thrust::device_vector<int> countvec(N);

    thrust::reduce_by_key(
            thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(N)),
            thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(N)) + (N*N),
            tmpvec.begin(),
            thrust::make_discard_iterator(),
            countvec.begin());

    thrust::copy(countvec.begin(), countvec.end(), counts.begin());

    return;
}
