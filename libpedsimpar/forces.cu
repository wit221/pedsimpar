#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_vector.h"
#include <thrust/device_vector.h>
#include <thrust/fill.h>

__global__ void kernelLookahead(double2 *e, double2 *p, double2 *v, int* tmp, int* count, int N) {
    // compute lookahead counts

    int other = blockIdx.x * blockDim.x + threadIdx.x;
    int me = blockIdx.y * blockDim.y + threadIdx.y;

    if (me >= N || other >= N) {
        return;
    }

    if (me != other) {
        const double pi = 3.14159265;
        double distancex = p[other].x - p[me].x;
        double distancey = p[other].y - p[me].y;
        double dist2 = distancex*distancex + distancey*distancey;
        if (dist2 < 400) {
            double at2v = atan2(-e[me].x, -e[me].y);
            double at2d = atan2(-distancex, -distancey);
            double at2v2 = atan2(-v[other].x, -v[other].y);
            double s = at2d - at2v;
            if (s > pi) s -= 2*pi;
            if (s < -pi) s += 2*pi;
            double vv = at2v - at2v2;
            if (vv > pi) vv -= 2*pi;
            if (vv < -pi) vv += 2*pi;
            if (abs(vv) > 2.5) {
                if ((s < 0) && (s > -0.3))
                    tmp[me*N + other] = -1;
                if ((s > 0) && (s < 0.3))
                    tmp[me*N + other] = 1;
            }
        }
    }

    __syncthreads();
    // now reduce
    // TODO make parallel
    if (other != 0) {
        return;
    }
    int total = 0;
    for (int i = 0; i < N; i++) {
        total += tmp[me*N + i];
    }
    count[me] = total;
}


__global__ void kernelSocial(double2 *p, double2 *v, double* tmpx, double* tmpy, double* socialx, double* socialy, int N) {
    int other = blockIdx.x * blockDim.x + threadIdx.x;
    int me = blockIdx.y * blockDim.y + threadIdx.y;

    if (me >= N || other >= N) {
        return;
    }

    if (me != other) {
        const double lambdaImportance = 2.0;
        const double gamma = 0.35;
        const double n = 2;
        const double n_prime = 3;
        const double m_pi = 3.14159265358979323846264338327950288;

        double2 diff = make_double2(p[other].x - p[me].x, p[other].y - p[me].y);
        if (diff.x*diff.x + diff.y*diff.y <= 64.0) {
            double difflength = hypot(diff.x, diff.y);
            double2 diffDirection = make_double2(diff.x / difflength, diff.y / difflength);
            double2 velDiff = make_double2(v[me].x - v[other].x, v[me].y - v[other].y);
            double2 interactionVector = make_double2(lambdaImportance * velDiff.x + diffDirection.x,
                                                     lambdaImportance * velDiff.y + diffDirection.y);
            double interactionLength = hypot(interactionVector.x, interactionVector.y);
            double2 interactionDirection = make_double2(interactionVector.x / interactionLength,
                                                        interactionVector.y / interactionLength);
            double angleThis = atan2(interactionDirection.y, interactionDirection.x);
            double angleOther = atan2(diffDirection.y, diffDirection.x);
            double theta = angleOther - angleThis;
            if (theta > M_PI) theta -= 2 * m_pi;
            else if(theta <= -M_PI) theta += 2 * m_pi;

            int thetaSign = (theta == 0) ? (0) : (theta / abs(theta));
            double B = gamma * interactionLength;

            double forceVelocityAmount = -exp(-difflength/B - (n_prime*B*theta)*(n_prime*B*theta));
            double forceAngleAmount = -thetaSign * exp(-difflength/B - (n*B*theta)*(n*B*theta));

            double2 forceVelocity = make_double2(forceVelocityAmount * interactionDirection.x,
                                                 forceVelocityAmount * interactionDirection.y);
            double2 forceAngle = make_double2(forceAngleAmount * -interactionDirection.y,
                                              forceAngleAmount * interactionDirection.x);

            tmpx[me*N + other] = forceVelocity.x + forceAngle.x;
            tmpy[me*N + other] = forceVelocity.y + forceAngle.y;
        }
    }

    __syncthreads();
    // now reduce
    // TODO make parallel
    if (other != 0) {
        return;
    }
    double2 total;
    total.x = 0.0;
    total.y = 0.0;
    for (int i = 0; i < N; i++) {
        total.x += tmpx[me*N + i];
        total.y += tmpy[me*N + i];
    }
    /*
    if (me == 517 && other == 0) {
        printf("inside %.10lf %.10lf\n", total.x, total.y);
    }
    */

    socialx[me] = total.x;
    socialy[me] = total.y;
}


void cudaLookaheadSocial(vector<Ped::Tagent*> &agents, vector<int> &counts, vector<double> &socialx, vector<double> &socialy) {
    int N = agents.size();
    thrust::host_vector<double2> pvec_host(N);
    thrust::host_vector<double2> vvec_host(N);
    thrust::host_vector<double2> evec_host(N);

    thrust::device_vector<int> tmpvec(N*N);
    thrust::fill(tmpvec.begin(), tmpvec.end(), 0);
    thrust::device_vector<int> countvec(N);

    int i = 0;
    for (std::vector<Ped::Tagent*>::iterator iter = agents.begin(); iter!=agents.end(); ++iter) {
        const Ped::Tagent* agent = *iter;
        Ped::Tvector op = agent->getPosition();
        Ped::Tvector ov = agent->getVelocity();
        Ped::Tvector oe = agent->getDesiredDirection();
        pvec_host[i] = make_double2(op.x, op.y);
        vvec_host[i] = make_double2(ov.x, ov.y);
        evec_host[i] = make_double2(oe.x, oe.y);
        i++;
    }

    thrust::device_vector<double2> pvec = pvec_host;
    thrust::device_vector<double2> vvec = vvec_host;
    thrust::device_vector<double2> evec = evec_host;

    dim3 blockDim(32, 32);
    dim3 gridDim((N + 32 - 1)/32, (N + 32 - 1)/32);
    kernelLookahead<<<gridDim, blockDim>>>(
        thrust::raw_pointer_cast(evec.data()),
        thrust::raw_pointer_cast(pvec.data()),
        thrust::raw_pointer_cast(vvec.data()),
        thrust::raw_pointer_cast(tmpvec.data()),
        thrust::raw_pointer_cast(countvec.data()),
        N
    );

    thrust::device_vector<double> tmpvec2x(N*N);
    thrust::device_vector<double> tmpvec2y(N*N);
    thrust::fill(tmpvec2x.begin(), tmpvec2x.end(), 0.0);
    thrust::fill(tmpvec2y.begin(), tmpvec2y.end(), 0.0);
    thrust::device_vector<double> socialvecx(N);
    thrust::device_vector<double> socialvecy(N);

    kernelSocial<<<gridDim, blockDim>>>(
        thrust::raw_pointer_cast(pvec.data()),
        thrust::raw_pointer_cast(vvec.data()),
        thrust::raw_pointer_cast(tmpvec2x.data()),
        thrust::raw_pointer_cast(tmpvec2y.data()),
        thrust::raw_pointer_cast(socialvecx.data()),
        thrust::raw_pointer_cast(socialvecy.data()),
        N
    );

    thrust::copy(countvec.begin(), countvec.end(), counts.begin());
    thrust::copy(socialvecx.begin(), socialvecx.end(), socialx.begin());
    thrust::copy(socialvecy.begin(), socialvecy.end(), socialy.begin());


    /*if (id == 7) {
        cerr << "cuda:";
        for (int i = 0; i < N; i++) {
            //cerr << result[i] << " ";
            cerr << pvec_host[i].x << " " << pvec_host[i].y << " " << vvec_host[i].x << " " << vvec_host[i].y << " " << result[i] << endl;
        }
        cerr << endl;
    }*/

    return;
}
