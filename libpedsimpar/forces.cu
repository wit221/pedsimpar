#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_vector.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

struct lookaheadForceFunctor : public thrust::binary_function<double2,double2,int>
{
    const double2 e;
    const double2 p;
    const double2 v;

    lookaheadForceFunctor(double2 _e, double2 _p, double2 _v) : e(_e), p(_p), v(_v) {}

    __host__ __device__
    float operator()(const double2& otherp, const double2& otherv) const { 
        const double pi = 3.14159265;
        double distancex = otherp.x - p.x;
        double distancey = otherp.y - p.y;
        double dist2 = distancex*distancex + distancey*distancey;
        if (dist2 < 400) {
            double at2v = atan2(-e.x, -e.y);
            double at2d = atan2(-distancex, -distancey);
            double at2v2 = atan2(-otherv.x, -otherv.y);
            double s = at2d - at2v;
            if (s > pi) s -= 2*pi;
            if (s < -pi) s += 2*pi;
            double vv = at2v - at2v2;
            if (vv > pi) vv -= 2*pi;
            if (vv < -pi) vv += 2*pi;
            if (abs(vv) > 2.5) {
                if ((s < 0) && (s > -0.3))
                    return -1;
                if ((s > 0) && (s < 0.3))
                    return 1;
            }
        }
        return 0;
    }
};

int cudaLookaheadCount(Ped::Tvector e, Ped::Tvector p, Ped::Tvector v, int id, const set<const Ped::Tagent*> &neighbors) {
    int N = neighbors.size() - 1;
    thrust::host_vector<double2> pvec_host(N);
    thrust::host_vector<double2> vvec_host(N);

    int i = 0;
    for (set<const Ped::Tagent*>::iterator iter = neighbors.begin(); iter!=neighbors.end(); ++iter) {
        const Ped::Tagent* other = *iter;

        // don't compute this force for the agent himself
        if (other->getid() == id) continue;
        Ped::Tvector op = other->getPosition();
        Ped::Tvector ov = other->getVelocity();
        pvec_host[i] = make_double2(op.x, op.y);
        vvec_host[i] = make_double2(ov.x, ov.y);
        i++;
    }

    thrust::device_vector<double2> pvec = pvec_host;
    thrust::device_vector<double2> vvec = vvec_host;
    thrust::device_vector<int> result(N);

    thrust::transform(pvec.begin(), vvec.begin(), pvec.end(), result.begin(),
        lookaheadForceFunctor(make_double2(e.x, e.y), make_double2(p.x, p.y), make_double2(v.x, v.y)));

    if (id == 7) {
        cerr << "cuda:";
        for (int i = 0; i < N; i++) {
            //cerr << result[i] << " ";
            cerr << pvec_host[i].x << " " << pvec_host[i].y << " " << vvec_host[i].x << " " << vvec_host[i].y << " " << result[i] << endl;
        }
        cerr << endl;
    }

    int lookaheadCount = thrust::reduce(result.begin(), result.end());
    return lookaheadCount;
}
