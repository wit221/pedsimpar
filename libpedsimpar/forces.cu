#include "hip/hip_runtime.h"
#include "ped_agent.h"
#include "ped_vector.h"
#include <thrust/device_vector.h>

__global__ void kernelLookahead(double2 *e, double2 *p, double2 *v, int* tmp, int* count, int N) {
    // compute lookahead counts

    int other = blockIdx.x * blockDim.x + threadIdx.x;
    int me = blockIdx.y * blockDim.y + threadIdx.y;

    if (me == other || me >= N || other >= N) {
        return;
    }

    const double pi = 3.14159265;
    double distancex = p[other].x - p[me].x;
    double distancey = p[other].y - p[me].y;
    double dist2 = distancex*distancex + distancey*distancey;
    if (dist2 < 400) {
        double at2v = atan2(-e[me].x, -e[me].y);
        double at2d = atan2(-distancex, -distancey);
        double at2v2 = atan2(-v[other].x, -v[other].y);
        double s = at2d - at2v;
        if (s > pi) s -= 2*pi;
        if (s < -pi) s += 2*pi;
        double vv = at2v - at2v2;
        if (vv > pi) vv -= 2*pi;
        if (vv < -pi) vv += 2*pi;
        if (abs(vv) > 2.5) {
            if ((s < 0) && (s > -0.3))
                tmp[me*N + other] = -1;
            if ((s > 0) && (s < 0.3))
                tmp[me*N + other] = 1;
        }
    }

    __syncthreads();
    // now reduce
    // TODO make parallel
    if (other != 0) {
        return;
    }
    int total = 0;
    for (int i = 0; i < N; i++) {
        total += tmp[me*N + i];
    }
    count[me] = total;
}

void cudaLookaheadCount(const vector<Ped::Tagent*> &agents, vector<int> &counts) {
    int N = agents.size();
    thrust::host_vector<double2> pvec_host(N);
    thrust::host_vector<double2> vvec_host(N);
    thrust::host_vector<double2> evec_host(N);

    thrust::device_vector<int> tmpvec(N*N);
    thrust::fill(thrust::device, tmpvec.begin(), tmpvec.end(), 0);
    thrust::device_vector<int> countvec(N);

    int i = 0;
    for (auto iter = agents.begin(); iter!=agents.end(); ++iter) {
        const Ped::Tagent* agent = *iter;
        Ped::Tvector op = agent->getPosition();
        Ped::Tvector ov = agent->getVelocity();
        Ped::Tvector oe = agent->getDesiredDirection();
        pvec_host[i] = make_double2(op.x, op.y);
        vvec_host[i] = make_double2(ov.x, ov.y);
        evec_host[i] = make_double2(oe.x, oe.y);
        i++;
    }

    thrust::device_vector<double2> pvec = pvec_host;
    thrust::device_vector<double2> vvec = vvec_host;
    thrust::device_vector<double2> evec = evec_host;

    dim3 blockDim(32, 32);
    dim3 gridDim((N + 32 - 1)/32, (N + 32 - 1)/32);
    kernelLookahead<<<gridDim, blockDim>>>(
        thrust::raw_pointer_cast(evec.data()),
        thrust::raw_pointer_cast(pvec.data()),
        thrust::raw_pointer_cast(vvec.data()),
        thrust::raw_pointer_cast(tmpvec.data()),
        thrust::raw_pointer_cast(countvec.data()),
        N
    );

    /*if (id == 7) {
        cerr << "cuda:";
        for (int i = 0; i < N; i++) {
            //cerr << result[i] << " ";
            cerr << pvec_host[i].x << " " << pvec_host[i].y << " " << vvec_host[i].x << " " << vvec_host[i].y << " " << result[i] << endl;
        }
        cerr << endl;
    }*/

    return 0;
}
